#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "cuda/cuda_data_compressor.h"
#include "types.h"

void gen_data(float * data, size_t data_size) {
    assert(data);
    srand(17);
    for (size_t i = 0; i < data_size; ++ i) {
        if (rand() % 2 == 0) {
            data[i] = 0;
        } else {
            data[i] = rand() % 10 - 5;
        }
    }
}

int main(int argc, char ** argv) {
    const size_t min_data_size = 1 * 1024 * 1024; 
    const size_t max_data_size = 128 * 1024 * 1024;
    const int count = 100;

    DataType * data_cpu = NULL;
    DataType * data_gpu = NULL;
    DataType * decompressed_data_cpu = NULL;
    data_cpu = new DataType [max_data_size];
    decompressed_data_cpu = new DataType [max_data_size];
    hipMalloc(&data_gpu, sizeof(DataType) * max_data_size);
    gen_data(data_cpu, max_data_size);

    for (size_t data_size = min_data_size; data_size <= max_data_size; 
            data_size *= 2) {
        printf("Checking the correctness (data size: %lu floats)...", data_size);
        hipMemcpy(data_gpu, data_cpu, sizeof(DataType) * data_size, 
                hipMemcpyHostToDevice);
        DataCompressor compressor(data_size);
        DataDecompressor decompressor(data_size);
        // verify the correctness first
        compressor.compress_data(data_gpu, true);
        DataType * compressed_data;
        size_t compressed_data_size;
        compressor.get_compressed_data(compressed_data, compressed_data_size);
        hipMemset(data_gpu, 0, sizeof(DataType) * data_size);
        decompressor.receive_compressed_data(
                [&](uint8_t * buff, size_t buff_size) {
                    assert(compressed_data_size <= buff_size);
                    memcpy(buff, compressed_data, compressed_data_size);
                    return compressed_data_size;
                }, true
                );
        decompressor.decompress_data(data_gpu);
        hipMemcpy(decompressed_data_cpu, data_gpu, sizeof(DataType) * data_size,
                hipMemcpyDeviceToHost);
        for (size_t i = 0; i < data_size; ++ i) {
            assert(decompressed_data_cpu[i] == data_cpu[i]);
        }
        printf("\tPassed\n");
    }

    hipMemcpy(data_gpu, data_cpu, sizeof(DataType) * max_data_size,
            hipMemcpyHostToDevice);
    for (size_t data_size = min_data_size; data_size <= max_data_size; 
            data_size *= 2) {
        printf("Benchmarking the performance (data size: %lu floats)...", data_size);
        DataCompressor compressor(data_size);
        DataDecompressor decompressor(data_size);
        auto start = std::chrono::system_clock::now();
        for (int i = 0; i < count; ++ i) {
            compressor.compress_data(data_gpu, false);
            DataType * compressed_data;
            size_t compressed_data_size;
            compressor.get_compressed_data(compressed_data, compressed_data_size);
            decompressor.receive_compressed_data(
                    [&](uint8_t * buff, size_t buff_size) {
                        assert(compressed_data_size <= buff_size);
                        hipMemcpy(buff, compressed_data, compressed_data_size, 
                                hipMemcpyDeviceToDevice);
                        return compressed_data_size;
                    }, false
                    );
            decompressor.decompress_data(data_gpu);
        }
        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        double t = elapsed_seconds.count() / count;
        double throughput = sizeof(DataType) * data_size / 1024. / 1024. / 1024. * 8. / t;
        printf("\tThroughput: %.3f Gbps\n", throughput);
    }
    return 0;
}



