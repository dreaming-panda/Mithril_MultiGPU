#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include"cuda/cuda_graph_parallel.h"
#include<hipDNN.h>
__global__ void Calculate_AccuracyMask(
DataType * cuda_acc_data,
DataType * cuda_output_data,
DataType * cuda_std_data,
int * mask,
int num_vertices,
int start,
int end,
int outputsize,
int ThreadNumber,
int BlockNumber,
int per_thread_nodes
){
    int nid_start = (blockIdx.x * ThreadNumber + threadIdx.x) * per_thread_nodes;
    int nid_end = nid_start + per_thread_nodes;
    if(nid_end >= num_vertices)nid_end = num_vertices;

    for(int i = nid_start; i < nid_end; ++i){
        cuda_acc_data[i] = 0.0f;
        int predict = 0;
        DataType * p = &cuda_output_data[i * outputsize];
        DataType * s = &cuda_std_data[i * outputsize];
        for(int j = 0; j < outputsize; ++j){
            if(p[j] > p[predict]){
                predict = j;
            }
        }
        if(s[predict] > 0.99 && mask[i] == 1 && i < end && i >= start)cuda_acc_data[i] = 1.0f;
    }
    
}
float CUDAGraphParallelEngine::LaunchCalculate_Accuracy_Mask(DataType * cuda_acc_data, DataType * cuda_output_data, DataType * cuda_std_data, int num_vertices, int outputsize, int type)
{   
    
    const int ThreadNumber = 512;
    const int BlockNumber =  (num_vertices + ThreadNumber - 1)/ThreadNumber;
    int per_thread_nodes = num_vertices / (ThreadNumber * BlockNumber) + 1;
    if(type == 0){

    Calculate_AccuracyMask<<<BlockNumber, ThreadNumber>>>(cuda_acc_data,cuda_output_data, cuda_std_data,gpu_training_mask_,num_vertices,start_vertex_, end_vertex_, outputsize, ThreadNumber, BlockNumber,per_thread_nodes);
    } else if (type == 1){
        Calculate_AccuracyMask<<<BlockNumber, ThreadNumber>>>(cuda_acc_data,cuda_output_data, cuda_std_data,gpu_valid_mask_,num_vertices,start_vertex_, end_vertex_, outputsize, ThreadNumber, BlockNumber,per_thread_nodes);
    } else if (type == 2){
         Calculate_AccuracyMask<<<BlockNumber, ThreadNumber>>>(cuda_acc_data,cuda_output_data, cuda_std_data,gpu_test_mask_,num_vertices,start_vertex_, end_vertex_, outputsize, ThreadNumber, BlockNumber,per_thread_nodes);
    }
    hipDeviceSynchronize();
    //hipdnnHandle_t cudnn_;
  //  DataType * d_hit_;
  //  DataType * d_inter_;
  //  AllocateCUDAMemory<DataType>(&d_hit_, 1, __FILE__, __LINE__);
  //  AllocateCUDAMemory<DataType>(&d_inter_, num_vertices, __FILE__, __LINE__);
  //  hipdnnCreate(&cudnn_);
  //  hipdnnReduceTensorDescriptor_t MeanDesc;
   // hipdnnCreateReduceTensorDescriptor(&MeanDesc);
    //hipdnnSetReduceTensorDescriptor(MeanDesc,HIPDNN_REDUCE_TENSOR_AVG,HIPDNN_DATA_FLOAT,HIPDNN_NOT_PROPAGATE_NAN,HIPDNN_REDUCE_TENSOR_NO_INDICES,HIPDNN_32BIT_INDICES);
    //hipdnnTensorDescriptor_t hit_descriptor;
    //hipdnnCreateTensorDescriptor(&hit_descriptor);
    hipdnnTensorDescriptor_t data_descriptor_;
    hipdnnCreateTensorDescriptor(&data_descriptor_);
    hipdnnSetTensor4dDescriptor(data_descriptor_, HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT, num_vertices, 1, 1, 1);
    //hipdnnSetTensor4dDescriptor(hit_descriptor, HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT, 1, 1, 1, 1);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipdnnReduceTensor(
        cudnn_,MeanDesc,nullptr,0,d_inter_, sizeof(DataType) * num_vertices,&alpha,
        data_descriptor_,cuda_acc_data,&beta,hit_descriptor,d_hit_
    );
    DataType acc = 0.0;
    CopyFromCUDADeviceToHost<DataType>(&acc, d_hit_, 1, __FILE__, __LINE__);
    if (type == 0){
        acc = acc * double(num_vertices) / double(ntrain);
    } else  if (type == 1){
        acc = acc * double(num_vertices) / double(nvalid);
    } else  if (type == 2){
        acc = acc * double(num_vertices) / double(ntest);
    }
    //hipdnnDestroy(cudnn_);
   // DeallocateCUDAMemory<DataType>(&d_hit_, __FILE__, __LINE__);
  //  DeallocateCUDAMemory<DataType>(&d_inter_, __FILE__, __LINE__);
    return acc;
}
__global__ void collect_mirrors_kernel(
    int mirror_vertices_number,
    int * mirror_vertices_list,
    int elements_per_vertex,
    DataType * src,
    DataType * dst
){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int s2 = mirror_vertices_list[idx] * elements_per_vertex; 
    int s1 = idx * elements_per_vertex;
    if(idx < mirror_vertices_number){
        memcpy(dst + s1, src + s2, elements_per_vertex * sizeof(DataType));
    }
}
__global__ void scatter_mirrors_kernel(
    int mirror_vertices_number,
    int * mirror_vertices_list,
    int elements_per_vertex,
    DataType * src,
    DataType * dst
){
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int s2 = mirror_vertices_list[idx] * elements_per_vertex; 
    int s1 = idx * elements_per_vertex;
    if(idx < mirror_vertices_number){
        memcpy(dst + s2, src + s1, elements_per_vertex * sizeof(DataType));
    }
}
void CUDAGraphParallelEngine::collect_mirrors(int mirror_vertices_number, int* mirror_vertices_list, int elements_per_vertex, DataType* src, DataType* dst)
{
    const int block_size = 1024;
    const int num_blocks = (mirror_vertices_number + block_size - 1) / block_size;
    collect_mirrors_kernel<<<num_blocks, block_size>>>(mirror_vertices_number, mirror_vertices_list, elements_per_vertex, src, dst);
    hipDeviceSynchronize();
}

void CUDAGraphParallelEngine::scatter_mirrors(int mirror_vertices_number, int* mirror_vertices_list, int elements_per_vertex, DataType* src, DataType* dst)
{
    const int block_size = 1024;
    const int num_blocks = (mirror_vertices_number + block_size - 1) / block_size;
    scatter_mirrors_kernel<<<num_blocks, block_size>>>(mirror_vertices_number, mirror_vertices_list, elements_per_vertex, src, dst);
    hipDeviceSynchronize();
}