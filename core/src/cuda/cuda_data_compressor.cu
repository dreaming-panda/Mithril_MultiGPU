#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

#include "types.h"
#include "cuda/cuda_data_compressor.h"
#include "cuda/cuda_executor.h"

#define BLOCK_SIZE 1024

DataCompressor::DataCompressor(size_t data_size) {
    data_compressed_ = false;
    compressed_data_on_cpu_ = false;
    // allocate the GPU buffer
    data_size_ = data_size;
    gpu_buff_size_ = (data_size / 32 + 1) * sizeof(uint32_t) 
        + sizeof(DataType) * data_size;
    assert(gpu_buff_size_ % sizeof(uint32_t) == 0);
    checkCUDA(hipMalloc(&gpu_buff_, gpu_buff_size_));
    gpu_bitmap_ = &gpu_buff_[0];
    gpu_non_zero_elements_ = (DataType*) &gpu_buff_[(data_size / 32 + 1) * sizeof(uint32_t)];
    // allocate the CPU buffer
    cpu_buff_size_ = gpu_buff_size_;
    cpu_buff_ = new uint8_t [cpu_buff_size_];
    assert(cpu_buff_);
}

DataCompressor::~DataCompressor() {
    // deallocate the buffers
    checkCUDA(hipFree(gpu_buff_));
    delete [] cpu_buff_;
}

struct non_zero_functor {
    __host__ __device__
        bool operator()(const DataType x) {
            return x != 0;
        }
};

__global__ void gen_bitmap_kernel(DataType * data, uint8_t * bitmap, size_t data_size) {
    size_t bitmap_idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t bitmap_size = data_size / 8 + 1;
    if (bitmap_idx < bitmap_size) {
        //printf("Bitmap idx: %lu / %lu, Data size\n", bitmap_idx, bitmap_size);
        size_t start = bitmap_idx * 8;
        size_t end = (bitmap_idx + 1) * 8;
        if (end > data_size) {
            end = data_size;
        }
        uint8_t v = 0;
        int offset = 0;
        for (size_t i = start; i < end; ++ i) {
            uint8_t mask = (data[i] == 0) ? 0: ((uint8_t) 1 << offset);
            offset ++;
            v = v ^ mask;
        }
        bitmap[bitmap_idx] = v;
    }
}

void DataCompressor::compress_data(DataType * data, bool send_to_cpu) {
    assert(! data_compressed_);

    size_t data_size = data_size_;
    uint8_t * bitmap = gpu_bitmap_;
    DataType * non_zero_elements = gpu_non_zero_elements_;

    // compress the data
    float * end_ptx = thrust::copy_if(
            thrust::cuda::par, data, data + data_size,
            non_zero_elements, non_zero_functor()
            );
    // get the number of non-zero elements
    uint64_t start_ptx_int = (uint64_t) non_zero_elements;
    uint64_t end_ptx_int = (uint64_t) end_ptx;
    assert(start_ptx_int <= end_ptx_int);
    assert((end_ptx_int - start_ptx_int) % sizeof(DataType) == 0);
    size_t num_non_zero_elements = (end_ptx_int - start_ptx_int) / sizeof(DataType);

    // generate the bitmap
    size_t bitmap_size = data_size / 8 + 1;
    int block_size = BLOCK_SIZE;
    int num_blocks = (bitmap_size + block_size - 1) / block_size;
    gen_bitmap_kernel<<<num_blocks, block_size>>>(data, bitmap, data_size);
    hipDeviceSynchronize();

    // calculate the size of the compressed data
    compressed_data_size_ = (data_size / 32 + 1) * sizeof(uint32_t)
        + sizeof(DataType) * num_non_zero_elements;

    if (send_to_cpu) {
        checkCUDA(hipMemcpy(cpu_buff_, gpu_buff_, compressed_data_size_,
                    hipMemcpyDeviceToHost));
    }

    data_compressed_ = true;
    compressed_data_on_cpu_ = send_to_cpu;
}

void DataCompressor::get_compressed_data(DataType * &buff, size_t &buff_size) {
    assert(data_compressed_);

    if (compressed_data_on_cpu_) {
        buff = (DataType*) cpu_buff_;
        buff_size = compressed_data_size_;
    } else {
        buff = (DataType*) gpu_buff_;
        buff_size = compressed_data_size_;
    }

    data_compressed_ = false;
    compressed_data_on_cpu_ = false;
}

DataDecompressor::DataDecompressor(size_t data_size) {
    data_size_ = data_size;
    compressed_data_set_ = false;
    compressed_data_on_cpu_ = false;

    gpu_buff_size_ = (data_size_ / 32 + 1) * sizeof(uint32_t)
        + sizeof(DataType) * data_size;
    checkCUDA(hipMalloc(&gpu_buff_, gpu_buff_size_));
    gpu_bitmap_ = &gpu_buff_[0];
    gpu_non_zero_elements_ = (DataType*) &gpu_buff_[(data_size / 32 + 1) * sizeof(uint32_t)];
    checkCUDA(hipMalloc(&gpu_data_decompression_index_, sizeof(uint32_t) * data_size));

    cpu_buff_size_ = gpu_buff_size_;
    cpu_buff_ = new uint8_t [cpu_buff_size_];
    assert(cpu_buff_);
}

DataDecompressor::~DataDecompressor() {
    checkCUDA(hipFree(gpu_buff_));
    checkCUDA(hipFree(gpu_data_decompression_index_));
    delete [] cpu_buff_;
}

void DataDecompressor::receive_compressed_data(std::function<size_t(uint8_t * buff)> recv_data, bool recv_on_cpu) {
    assert(! compressed_data_set_);
    compressed_data_on_cpu_ = recv_on_cpu;

    if (recv_on_cpu) {
        compressed_data_size_ = recv_data(cpu_buff_);
    } else {
        compressed_data_size_ = recv_data(gpu_buff_);
    }

    compressed_data_set_ = true;
}

__global__ void gen_decompression_index_kernel(
        uint8_t * bitmap, uint32_t * decompression_index, size_t data_size
        ) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < data_size) {
        size_t bitmap_idx = idx / 8;
        size_t bitmap_offset = idx % 8;
        uint8_t mask = (uint8_t) 1 << bitmap_offset;
        uint8_t is_not_zero = bitmap[bitmap_idx] & mask;
        decompression_index[idx] = is_not_zero ? 1: 0;
    }
}

__global__ void decompress_data_kernel(
        uint32_t * decompression_index, DataType * non_zero_elements, DataType * decompressed_data, uint8_t * bitmap, size_t data_size
        ) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < data_size) {
        size_t bitmap_idx = idx / 8;
        size_t bitmap_offset = idx % 8;
        uint8_t mask = (uint8_t) 1 << bitmap_offset;
        uint8_t is_not_zero = bitmap[bitmap_idx] & mask;
        DataType data = non_zero_elements[decompression_index[idx]];
        decompressed_data[idx] = is_not_zero ? data: 0.;
    }
}

void DataDecompressor::decompress_data(DataType * data) {
    assert(compressed_data_set_);

    if (compressed_data_on_cpu_) {
        checkCUDA(hipMemcpy(gpu_buff_, cpu_buff_, compressed_data_size_,
                    hipMemcpyHostToDevice));
    }

    size_t data_size = data_size_;

    assert(compressed_data_size_ >= (data_size / 32 + 1) * sizeof(uint32_t));
    size_t num_non_zero_elements = compressed_data_size_ - (data_size / 32 + 1) * sizeof(uint32_t);
    assert(num_non_zero_elements % sizeof(DataType) == 0);
    num_non_zero_elements /= sizeof(DataType);
    uint8_t * bitmap = gpu_bitmap_;
    DataType * non_zero_elements = gpu_non_zero_elements_;
    uint32_t * decompression_index = gpu_data_decompression_index_;

    int block_size = BLOCK_SIZE;
    int num_blocks = (data_size + block_size - 1) / block_size;
    gen_decompression_index_kernel<<<num_blocks, block_size>>>(bitmap, decompression_index, data_size);
    hipDeviceSynchronize();
    thrust::exclusive_scan(thrust::cuda::par, decompression_index, decompression_index + data_size, decompression_index);
    decompress_data_kernel<<<num_blocks, block_size>>>(decompression_index, non_zero_elements, data, bitmap, data_size); 
    hipDeviceSynchronize();

    compressed_data_set_ = false;
    compressed_data_on_cpu_ = false;
}



